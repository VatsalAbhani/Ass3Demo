
#include <hip/hip_runtime.h>
#include <stdio.h>
#define TILE_SIZE 16 

__global__ void mysgemm(int m, int n, int k, const float *A, const float *B, float* C) {
    int bIdx = blockIdx.x;
    int bIdy = blockIdx.y;

    int tIdx = threadIdx.x;
    int tIdy = threadIdx.y;

    int Row = bIdy * blockDim.y + tIdy;
    int Col = bIdx * blockDim.x + tIdx;

   __shared__ float As[TILE_SIZE][TILE_SIZE];
   __shared__ float Bs[TILE_SIZE][TILE_SIZE]; 
 
    float Result = 0.0;

    for(int ph = 0; ph < (k-1) / TILE_SIZE + 1; ++ph) {
	int globalRow = ph * TILE_SIZE + tIdy;
        int globalCol = ph * TILE_SIZE + tIdx;

	if (Row < m && globalCol < k)
	As[tIdy][tIdx] = A[Row * k + globalCol]; 
        else
	As[tIdy][tIdx] = 0.0; 

	if(globalRow < k && Col < n)
	Bs[tIdy][tIdx] = B[globalRow * n + Col];
        else
	Bs[tIdy][tIdx] = 0.0; 

        __syncthreads();

        for (int i = 0; i < TILE_SIZE; ++i) 
		Result += As[tIdy][i] * Bs[i][tIdx];
        __syncthreads();
    }

   if(Row < m && Col < n)
    C[Row * n + Col] = Result; 
}

void basicSgemm(int m, int n, int k, const float *A, const float *B, float *C)
{
    // Initialize thread block and kernel grid dimensions ---------------------

    const unsigned int BLOCK_SIZE = TILE_SIZE;
	
    /*************************************************************************/
    //INSERT CODE HERE
   dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid((n - 1) / dimBlock.x + 1, (m - 1) / dimBlock.y + 1);
    /*************************************************************************/

    // Invoke CUDA kernel -----------------------------------------------------

    /*************************************************************************/
    //INSERT CODE HERE
	 mysgemm<<<dimGrid, dimBlock>>>(m, n, k, A, B, C);
    /*************************************************************************/
}



